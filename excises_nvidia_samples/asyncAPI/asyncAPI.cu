#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <julia.h>
#include <hip/hip_runtime.h>

__global__ void vec_add(float *d_A,float *d_B,float *d_C, int len)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int idx = threadIdx.x;
    if ( idx<len )
    {
        d_C[idx] = d_A[idx] + d_B[idx];

        /*
        printf("d_A[%d] is %f\n",idx,d_A[idx]);
        printf("d_B[%d] is %f\n",idx,d_B[idx]);

        printf("d_C[%d] is %f\n",idx,d_C[idx]);
        */
    }
}

int main(void)
{
    // init julia
    jl_init("/home/guo/julia/usr/lib");
    jl_eval_string("println(\"julia: julia init successfully\")");

    printf("sizeof(double) is %d\n", (int)sizeof(double));
    printf("sizeof(int) is %d\n", (int)sizeof(int));
    printf("sizeof(float) is %d\n", (int)sizeof(float));

    float *h_A,*h_B,*h_C;
    // vectAdd and length(vec) = 10000
    int block_len = 1024;
    int num_block = 1024*16;
    int len = block_len*num_block;
    size_t size = len*sizeof(float);
    // malloc host mem use hipHostMalloc
    hipHostMalloc((void **)&h_A, size);
    hipHostMalloc((void **)&h_B, size);
    hipHostMalloc((void **)&h_C, size);
//    h_A = (float *)malloc(size);
//    h_B = (float *)malloc(size);
//    h_C = (float *)malloc(size);

    //init vector A,B,C
    for (int i=0;i<len;i++)
    {

//        memset((h_A+i),i,sizeof(float));
//        memset((h_B+i),i,sizeof(float));
        h_A[i] = i;
        h_B[i] = 2*i;
        h_C[i] = 0;
    }
//    memset(h_C,0,size);
    jl_eval_string("println(\"julia: host A,B,C init successfully\")");
   
    // alloc device mem
    float *d_A,*d_B,*d_C;
    hipMalloc((void **)&d_A,size);
    hipMalloc((void **)&d_B,size);
    hipMalloc((void **)&d_C,size);

    /*
    for (int i=0;i<len;i++)
    {
        printf("h_B[%d] is %f\n",i,h_B[i]);
    }
    */

    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // memcpy ==> vec_add<<<>>> ==> memcpy #TODO stream and event async parallel maxthroughtput
    // data transfer from host to device
    hipDeviceSynchronize();
    hipEventRecord(start,0);
    hipMemcpyAsync(d_A,h_A,size,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_B,h_B,size,hipMemcpyHostToDevice,0);

    dim3 dimGrid(num_block,1);
    dim3 dimBlock(block_len,1);
    vec_add<<<dimGrid,dimBlock,0,0>>>(d_A,d_B,d_C,len);
//    hipDeviceSynchronize();

    hipMemcpyAsync(h_C,d_C,size,hipMemcpyDeviceToHost,0); // hipMemcpy is a synchronize function
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    float gpu_time;
    hipEventElapsedTime(&gpu_time,start,stop);
    printf("gpu_time is %f\n",gpu_time);
    // check result 
    bool ret = true;
    for (int i=0;i<len;i++)
    {
        if (h_C[i] != 3*i)
        {   
           // printf("h_C[%d] is %f",i,h_C[i]);
            ret = false;
        }
    }
    
    ret ? printf("result is passed\n") : printf("result is failed\n");

    //Free objects
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    jl_atexit_hook(0);
    hipDeviceReset();
    return 0;
}
