#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>

#define MAX_EPSILON_ERROR 5e-3f

const char *imageFilename = "lena_bw.pgm";
const char *refFilename = "ref_rotated.pgm";
const char *sampleName = "simpleTexture";
const float angle = 0.5f;

texture<float, 2, hipReadModeElementType> tex;

__global__ void transform_kernel(float *outputData, int width, int height, float theta)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = (float)x - (float)width/2;
    float v = (float)y - (float)height/2;
    float tu = u*cosf(theta) - v*sinf(theta);
    float tv = v*cosf(theta) + v*sinf(theta);

    tu /= (float)width;
    tv /= (float)height;

    outputData[y*width+x] = tex2D(tex, tu+0.5f, tv+0.5f);
}

void run_test(int argc, char **argv);

int main(int argc, char **argv)
{
    run_test(argc, argv);

    hipDeviceReset();
    return 0;
}

void run_test(int argc, char **argv)
{
    int devID = 1;
    
    float *h_data = NULL;
    unsigned int width, height;
    const char *image_path = 
        "/home/guo/haplox/GPU/NVIDIA_CUDA-7.0_Samples/0_Simple/simpleTexture/data/lena_bw.pgm"

    sdkLoadPGM(image_path, &h_data, &width, &height);

    unsigned int size = width * height * sizeof(float);

    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    float *h_data_ref = (float *)malloc(size);
    const char *ref_path = 
         "/home/guo/haplox/GPU/NVIDIA_CUDA-7.0_Samples/0_Simple/simpleTexture/data/ref_rotated.pgm"
    sdkLoadPGM(&ref_path, &h_data_ref,&width, &height);

    float *d_data = NULL;
    hipMalloc((void **)&d_data, size);

    hipChannelFormatDesc channelDesc = 
        hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
    hipArray *cuArray;
    hipMallocArray(&cuArray,&channelDesc,width,height);
    hipMemcpyToArray(cuArray,0,0,h_data,size,hipMemcpyHostToDevice);
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;
    hipBindTextureToArray(tex,cuArray,channelDesc);

    dim3 dim_block(8,8);
    dim3 dim_grid(width/dim_block.x, height/dim_block.y);

    transform_kernel<<<dim_grid, dim_block>>>(d_data, width, height, angle);
    hipDeviceSynchronize();
    
    float *h_output_data = (float *) malloc(size);
    hipMemcpy(h_output_data, d_data, size, hipMemcpyDeviceToHost);
    printf("%d\t%d\t%d\n", h_output_data[0],h_output_data[1],h_output_data[2]);

    hipFree(d_data);
    hipFreeArray(cuArray);
    free(image_path);
    free(ref_path);
}
