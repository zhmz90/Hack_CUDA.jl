
#include <hip/hip_runtime.h>
#include <iostream>

const int manualBlockSize = 32;


__global__ void square(int *array, int len)
{
    int gtid = blockDim.x*blockIdx.x + threadIdx.x;
    if (gtid < len)
        array[gtid] *= array[gtid];

}

// active warps / maximum warps per SM
static double reportPotentialOccupancy(void *kernel, int blockSize, size_t dynamicSMem)
{
    int device;
    hipDeviceProp_t prop;

    int numBlocks;
    int activeWarps;
    int maxWarps;

    double occupancy;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, kernel, blockSize, dynamicSMem);

    activeWarps = numBlocks*blockSize / prop.warpSize;
    maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

    occupancy = (double)activeWarps / maxWarps;

    return occupancy;
}

static int  launchConfig(int *array, int arrayCount, bool automatic)
{
    int blockSize;
    int minGridSize;
    int gridSize;
    size_t dynamicSMemUsage = 0;

    hipEvent_t start;
    hipEvent_t end;

    float elapsedTime;

    double potentialOccupancy;

    hipEventCreate(&start);
    hipEventCreate(&end);
    
    if (automatic)
    {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void *)square, 
                dynamicSMemUsage, arrayCount);
        std::cout << "Suggested block size:"<< blockSize << std::endl;
    }
    else
    {
        blockSize = manualBlockSize;
    }
    
    gridSize = (arrayCount + blockSize - 1)/blockSize;

    hipEventRecord(start);
    square<<<gridSize, blockSize, dynamicSMemUsage>>>(array, arrayCount);
    hipEventRecord(end);
        
    hipDeviceSynchronize();

    potentialOccupancy = reportPotentialOccupancy((void *)square, blockSize, dynamicSMemUsage);

    std::cout << "potential occupancy:"<<potentialOccupancy * 100 << "%" <<std::endl;

    hipEventElapsedTime(&elapsedTime, start, end);

    std::cout<< "Elapsed time:" <<elapsedTime << "ms" << std::endl;

    return 0;
}

static int test(bool automaticLaunchConfig, const int count = 1000000)
{
    int *array;
    int *dArray;
    int size = count * sizeof(int);

    array = new int[count];

    for (int i=0;i<count;i++)
    {
        array[i] = i;
    }

    hipMalloc(&dArray,size);
    hipMemcpy(dArray, array, size, hipMemcpyHostToDevice);

    for (int i=0; i< count; i++)
    {
        array[i] = 0;
    }
    launchConfig(dArray,count,automaticLaunchConfig);
    
    hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost);
    hipFree(dArray);

    for (int i=0;i<count;i++)
    {
        if (array[i] != i*i)
        {
            std::cout << "element" << i <<" expected "<< i*i <<" actual "<<array[i]<<std::endl;
            return  1;
        }
    }
    hipDeviceReset();
    delete[] array;

    return 0;
}

int main()
{
    int status;
    std::cout << "[ Manual configuration with "<<manualBlockSize
              << " threads per block ]" << std::endl;

    test(false);
    test(true);

    return 0;
}
