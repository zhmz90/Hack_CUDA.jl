#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/mman.h> // mmap() munmap()

#include <julia.h>
#include <hip/hip_runtime.h>


#define MEMORY_ALIGNMENT 4096
#define ALIGN_UP(x, size) ( ((size_t)x+(size-1)&(~(size-1))) ) // ????

/*
const char *sEventSyncMethod[] =
{
    "hipEventDefault",
    "hipEventBlockingSync".
    "hipEventDisableTiming",
    "NULL"
};

const char *sDeviceSyncMethod[] = 
{
    "hipDeviceScheduleAuto",
    "hipDeviceScheduleSpin",
    "hipDeviceScheduleYield",
    "INVALID",
    "hipDeviceScheduleBlockingSync",
    NULL
};
*/

// why int *factor not int?
__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=0; i<num_iterations; i++)
    {
        g_data[idx] += *factor; // non-coalesced 
    }

}

// a[] must all be c
bool correct_data(int *a, const int n, const int c)
{   
    for (int i=0;i<n;i++)
    {
        if (a[i]!=c)
        {
            printf("correct_data is wrong\n");
            return false;
        }
    }
    return true;
}


// ??? what is the difference between mmap+hipHostRegister
inline void alloc_host_mem(bool b_pin_generic_mem, int **pp_a, int **pp_aligned_a, int nbytes)
{
    if (b_pin_generic_mem)
    {
        *pp_a = (int *)mmap(NULL, (nbytes+MEMORY_ALIGNMENT), PROT_READ|PROT_WRITE, 
                            MAP_PRIVATE|MAP_ANON, -1, 0);

        *pp_aligned_a = (int *)ALIGN_UP(*pp_a, MEMORY_ALIGNMENT);
    
        hipHostRegister(*pp_aligned_a, nbytes, hipHostRegisterMapped);
    }
    else
    {
        hipHostMalloc((void **)pp_a, nbytes);
        *pp_aligned_a = *pp_a;
    }

}
inline void free_host_mem(bool b_pin_generic_mem, int **pp_a, int **pp_aligned_a, int nbytes)
{
    if (b_pin_generic_mem)
    {
        hipHostUnregister(*pp_aligned_a);
        munmap(*pp_a, nbytes);
    }
    else
    {
        hipHostFree(*pp_a);
    }
}


int main(void)
{
    jl_init("/home/guo/julia/usr/lib");

    int dev_count;
    hipGetDeviceCount(&dev_count);

    jl_function_t *func = jl_get_function(jl_base_module,"println");
    jl_value_t* num_dev = jl_box_int32(dev_count);
    jl_call1(func,num_dev);

    int cuda_device = 0;
    int nstreams = 4;
    int nreps = 10;
    int n = 16*1024*1024;
    int nbytes = n * sizeof(int);
    dim3 dim_block, dim_grid;
    float elapsed_time, time_memcpy, time_kernel;
    float scale_factor = 1.0f;

    bool b_pin_generic_mem = true;

    int device_sync_method = hipDeviceScheduleBlockingSync;

    int niterations=5;

    hipSetDevice(cuda_device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, cuda_device);
    printf("%s:%d.%d\n",deviceProp.name,deviceProp.major,deviceProp.minor);
    printf("deviceProp.canMapHostMemory:%d\n",(int)deviceProp.canMapHostMemory);

    scale_factor = 32.0f / 3072;

    n = (int)rint((float)n/scale_factor);

    printf("scale_factor: %1.4f\n",1.0f/scale_factor);
    printf("array_size: %d\n\n",n);

    //using CPU/GPU device sync, blocking sync to reduce CU uage
    hipSetDeviceFlags(device_sync_method | hipDeviceMapHost );

    // allocate host mem
    int h_c = 5;
    int *h_a = 0;
    int *h_aligned_a = 0;

    alloc_host_mem(b_pin_generic_mem,&h_a,&h_aligned_a,nbytes);

    // alloc device mem
    int *d_a = 0, *d_c = 0;
    hipMalloc((void **)&d_a, nbytes);
    hipMalloc((void **)&d_c, nbytes);
    hipMemcpy(d_c, &h_c, sizeof(int),hipMemcpyHostToDevice);

    hipStream_t *streams = (hipStream_t *)malloc(nstreams*sizeof(hipStream_t));
    
    for (int i=0;i<nstreams;i++)
    {
        hipStreamCreate(&(streams[i])); //hipStreamCreate((streams+i));
    }

    // event handles 
    // use blocking sync
    hipEvent_t start_event, stop_event;
    int event_flags = hipEventBlockingSync;
    hipEventCreateWithFlags(&start_event,event_flags);
    hipEventCreateWithFlags(&stop_event,event_flags);

    hipEventRecord(start_event,0);
    hipMemcpyAsync(h_aligned_a,d_a,nbytes,hipMemcpyDeviceToHost,streams[0]);
    hipEventRecord(stop_event,0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&time_memcpy,start_event,stop_event);
    printf("memcpy:\t%.2f\n",time_memcpy);
    
    // time kernel
    dim_block = dim3(512,1);
    dim_grid  = dim3(n/dim_block.x,1);
    hipEventRecord(start_event,0);
    init_array<<<dim_grid, dim_block, 0, streams[0]>>>(d_a,d_c,niterations);
    hipEventRecord(stop_event,0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&time_kernel,start_event,stop_event);
    printf("kernel:\t\t%.2f\n",time_kernel);

    //////////////////////////////////////
    printf( "#################time non-stream exec##############\n");
    dim_block = dim3(512,1);
    dim_grid  = dim3(n/dim_block.x,1); 
    hipEventRecord(start_event,0);

    for (int k=0; k<nreps;k++)
    {
        init_array<<<dim_grid,dim_block>>>(d_a,d_c,niterations);
        hipMemcpy(h_aligned_a, d_a, nbytes,hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_event,0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);
    printf("non-stream:\t%.2f\n",elapsed_time/nreps);

    printf( "\n#################time with nstreams streams exec##############\n");
    dim_block = dim3(512,1);
    dim_grid  = dim3(n/dim_block.x,1);
    memset(h_aligned_a,255,nbytes);
    hipMemset(d_a,0,nbytes);
    hipEventRecord(start_event,0);
    
    for (int k=0;k<nreps;k++)
    {
        for (int i=0;i<nstreams;i++)
        {
            init_array<<<dim_grid,dim_block,0,streams[i]>>>(d_a+i*n/nstreams,d_c,niterations);
        }
        
        for (int i=0;i<nstreams;i++)
        {
            hipMemcpyAsync(h_aligned_a+i*n/nstreams,d_a+i*n/nstreams,nbytes/nstreams, hipMemcpyDeviceToHost,streams[i]);
        }
    }
    hipEventRecord(stop_event,0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time,start_event,stop_event);
    printf("time with nstreams streams exec %f\n", elapsed_time/nreps); 

    for (int i=0;i<nstreams;i++)
    {
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    free_host_mem(b_pin_generic_mem, &h_a, &h_aligned_a, nbytes);

    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();
    jl_atexit_hook(0);

    return 0;
}
