#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "helper_h.h"

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_value; // may overflow

}

bool correct_output(int *data, const int n, const int x)
{
    for (int i=0;i<n;i++)
    {
        if (data[i] != x)
        {
            printf("error in correct_output\n");
            return false;
        }
    }
    return true;
}

int main(int argc, char *argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;
//    devID = findCudaDevice(argc, (char **)argv);
    devID = 0;
    check_error(hipGetDeviceProperties(&deviceProps, devID));

    printf("CUDA device [%s] \n", deviceProps.name);

    int n = 16*1024*1024;
    int nbytes = n*sizeof(int);

    int value = 26;

    // alloc host mem
    int *h_a = 0;
    check_error(hipHostMalloc((void **)&h_a,nbytes));
    memset(h_a, 0, nbytes);

    // alloc devie
    int *d_a = 0;
    check_error(hipMalloc((void **)&d_a, nbytes));
    check_error(hipMemset(d_a,0,nbytes));

    dim3 dim_block(512,1);
    dim3 dim_grid(n/dim_block.x,1);

    hipEvent_t start, stop;
    check_error(hipEventCreate(&start));
    check_error(hipEventCreate(&stop));
/*
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
*/
    check_error(hipDeviceSynchronize());

    float gpu_time = 0.0f;
//    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a,h_a,nbytes,hipMemcpyHostToDevice, 0);
    increment_kernel<<<dim_grid,dim_block>>>(d_a, value);
    hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
//    sdkStopTimer(&timer);

    // hipEventQuery 
    float counter=0;
    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }
    check_error(hipEventElapsedTime(&gpu_time, start, stop));

    printf("gpu_time is %f\n",gpu_time);
//   printf("sdkGetTimerValue(&timer is %f\n)",sdkGetTimerValue(&timer))
    printf("counter is %f\n",counter);

    bool cpu_gpu = correct_output(h_a, n ,value);

    check_error(hipEventDestroy(start));
    check_error(hipEventDestroy(stop));
    check_error(hipHostFree(h_a));
    check_error(hipFree(d_a));

    hipDeviceReset();
    exit(cpu_gpu ? EXIT_SUCCESS : EXIT_FAILURE);
}
