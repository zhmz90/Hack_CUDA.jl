
#include <hip/hip_runtime.h>
extern "C"
{
    __global__ void vadd(const float *a, const float *b, float *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        c[i] = a[i] + b[i];
    
    }


}
